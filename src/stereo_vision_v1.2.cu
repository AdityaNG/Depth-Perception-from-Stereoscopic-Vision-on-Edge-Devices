#include "hip/hip_runtime.h"
//#include <curl/curl.h>
#include <exception>
#include <iostream>
#include <opencv4/opencv2/highgui.hpp>
#include <vector>
#include <thread> 
#include <stdlib.h>
#include <fstream>
#include <ctime>
#include <opencv2/opencv.hpp>
#include <opencv2/calib3d.hpp>
#include <string.h>
#include <math.h>
#include <popt.h>
#include <future>
#include <omp.h>

#include "yolo/yolo.hpp"
#include "elas_cuda_openmp/elas.h"
#include "elas_cuda_openmp/elas_gpu.h"
#include "graphing/graphing.h"
#include "cleanup/cleanup.hpp"
#include "bayesian/bayesian.h"


#define GL_GLEXT_PROTOTYPES
#ifdef __APPLE__
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif

std::vector<OBJ> obj_list, pred_list;

using namespace cv;
using namespace std;

#define shrink_factor 2 // Modify to change the image resize factor

#define start_timer auto start = chrono::high_resolution_clock::now();  
#define end_timer(var)\
  auto end = chrono::high_resolution_clock::now();\
  double time_taken =  chrono::duration_cast<chrono::nanoseconds>(end - start).count();\
  time_taken *= 1e-9;\
  var = time_taken;                                             

//////////////////////////////////////// Globals ///////////////////////////////////////////////////////
Mat XR, XT, Q, P1, P2;
Mat R1, R2, K1, K2, D1, D2, R;
Mat lmapx, lmapy, rmapx, rmapy;
Mat left_img_OLD, right_img_OLD, dmapOLD;
Vec3d T;
FileStorage calib_file;

Size out_img_size;
Size calib_img_size;
int calib_width = 1242, calib_height = 375,
    out_width = 1242/shrink_factor, out_height = 375/shrink_factor;

const char* kitti_path;
const char* calib_file_name = "calibration/kitti_2011_09_26.yml";

double pc_t = 0, yd_t = 0, t_t = 0; // For calculating timings

int video_mode = 0;
int debug = 0;
int draw_points = 0;
int frame_skip = 1;
int play_video = 0;

// Cuda globals
double *d_XT, *d_XR, *d_Q;
uchar *d_dmap; // Disparity map needs to be pushed to GPU
double3 *points; // Holds the coordinates of each pixel in 3D space
double3 *d_points;
uchar4 *color = NULL;
hipStream_t s1;
const dim3 blockSize(32, 32, 1);
const dim3 gridSize((out_width / blockSize.x) + 1, (out_height / blockSize.y) + 1, 1);
////////////////////////////////////////////////////////////////////////////////////////////////////////
/*
Elas::parameters param(Elas::MIDDLEBURY);
param.postprocess_only_left = true;
ElasGPU elas(param);*/
  //Elas::parameters param(Elas::ROBOTICS);
  //Elas::parameters param;
  

  //param.postprocess_only_left = false;
  
  //Elas elas(param);
  

void cudaInit(){
  // Cuda Init
  hipMalloc(&d_XT, sizeof(double) * 3);
  hipMalloc(&d_XR, sizeof(double) * 9);
  hipMalloc(&d_Q, sizeof(double) * 16);
  hipMalloc(&d_dmap, sizeof(uchar) * out_width * out_height);
  hipMalloc(&d_points, sizeof(double3) * out_width * out_height);
  points = (double3*)malloc(sizeof(double3) * out_width * out_height);
  hipStreamCreate(&s1);  
  printf("CUDA Init done\n");
}

void clean(){
  // Cuda Cleanup
  printf("Exitting the program.....\n");
  destroyAllWindows();
  free(points);
  hipStreamDestroy(s1);
  hipFree(d_XR);
  hipFree(d_XT);
  hipFree(d_Q);
  hipFree(d_points);
  hipFree(d_dmap);
  //elas.cudaDest();
  exit(0);
}

int constrain(int a, int lb, int ub) {
  if (a<lb)
    return lb;
  else if (a>ub)
    return ub;
  else
    return a;
}

/*
 * Function:  composeRotationCamToRobot 
 * --------------------
 * Given a (x,y,z) rotation params, a corresponding 3D rotation matrix is generated
 *
 *  float x: The x rotation
 *  float y: The y rotation
 *  float z: The z rotation
 *  returns: Mat The 3D rotation matrix
 *
 */
Mat composeRotationCamToRobot(float x, float y, float z) {
  Mat X = Mat::eye(3, 3, CV_64FC1);
  Mat Y = Mat::eye(3, 3, CV_64FC1);
  Mat Z = Mat::eye(3, 3, CV_64FC1);
  
  X.at<double>(1,1) = cos(x);
  X.at<double>(1,2) = -sin(x);
  X.at<double>(2,1) = sin(x);
  X.at<double>(2,2) = cos(x);

  Y.at<double>(0,0) = cos(y);
  Y.at<double>(0,2) = sin(y);
  Y.at<double>(2,0) = -sin(y);
  Y.at<double>(2,2) = cos(y);

  Z.at<double>(0,0) = cos(z);
  Z.at<double>(0,1) = -sin(z);
  Z.at<double>(1,0) = sin(z);
  Z.at<double>(1,1) = cos(z);
  
  return Z*Y*X;
}

/*
 * Function:  composeTranslationCamToRobot 
 * --------------------
 * Given a (x,y,z) translation params, a corresponding 3D tranlation matrix is generated
 *
 *  float x: The x translation
 *  float y: The y translation
 *  float z: The z translation
 *  returns: Mat The 3D tranlation matrix
 *
 */
Mat composeTranslationCamToRobot(float x, float y, float z) {
  return (Mat_<double>(3,1) << x, y, z);
}

/*
 * Function:  publishPointCloud 
 * --------------------
 * Given a disparity map, a corresponding 3D point cloud can be easily constructed. 
 * The Q matrix stored in the calibration file is used for this conversion. 
 * The reconstruction is mathematically expressed by the following matrix equation.
 *
 *               [  [1 0 0      -Cx         ];
 * (X,Y,Z,W)^T =    [0 1 0      -Cy         ];     . [x y d(x,y) 1]^T
 *                  [0 0 0      f           ]; 
 *                  [0 0 -1/Tx  (Cx-C'x)/Tx ]; ] 
 *
 * d(x,y)  is the disparity of a point (x,y) in the left image
 * The 4X4 matrix dentoes the Q matrix 
 *
 * The point cloud generated is in the reference frame of the left camera. 
 * Hence a transformation (XR, XT) is applied to transform the point cloud into a different reference frame 
 * (as required by the user). The transformation equation is as follows
 * PB = R × PA + T
 *
 * Q Matrix
 * [1, 0, 0,                  -339.7460250854492;
 *  0, 1, 0,                  -110.0997492116292;
 *  0, 0, 0,                  455.4106857822576;
 *  0, 0, 1.861616069957151,  -0]
 *
 *  Mat& img_left: The input left image - set of points (x, y)
 *  Mat& dmap: input disparity map d(x, y)
 *  returns: void
 *
 */


 __global__ void parallel(const uchar *dmap, double3 *points, int rows, int cols, const double *d_XT, const double *d_XR, const double *d_Q){
  // Calculating the coordinates of the pixel
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// To prevent trying to access data outside the image
	if (x >= cols || y >= rows)
        return;
    
  int pixelPosition = y * cols + x;
  uchar d = dmap[pixelPosition];
  //if(d < 2) return;

  double pos[4];
  for(int j = 0; j<4; j++) pos[j] = d_Q[4*j + 0]*x + d_Q[4*j + 1]*y + d_Q[4*j + 2]*d + d_Q[4*j + 3];
    
  double X = pos[0] / pos[3];
  double Y = pos[1] / pos[3];
  double Z = pos[2] / pos[3];

  double point[3];
  for(int j = 0; j<3; j++) point[j] = d_XR[3*j + 0]*X + d_XR[3*j + 1]*Y + d_XR[3*j + 2]*Z + d_XT[j];
    
  points[pixelPosition] = make_double3(point[0], point[1], point[2]);
}

 void publishPointCloud(Mat& img_left, Mat& dmap) { 
  if (img_left.empty() || dmap.empty()) {
    printf("(empty)\t");
    return;
  }
  if (debug == 1) {
    XR = composeRotationCamToRobot(1.3 ,-3.14,1.57);
    XT = composeTranslationCamToRobot(0.0,0.0,0.28);
    cout << "Rotation matrix: " << XR << endl;
    cout << "Translation matrix: " << XT << endl;
  }

  start_timer; 
  
  hipMemcpyAsync(d_dmap, dmap.data, sizeof(uchar) * out_width * out_height, hipMemcpyHostToDevice, s1);
  hipMemcpy(d_XT, XT.data, sizeof(double) * 3, hipMemcpyHostToDevice);
  hipMemcpy(d_XR, XR.data, sizeof(double) * 9, hipMemcpyHostToDevice);
  hipMemcpy(d_Q, Q.data, sizeof(double) * 16, hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  parallel <<<gridSize, blockSize, 0, s1>>> (d_dmap, d_points, out_height, out_width, d_XT, d_XR, d_Q);

  hipDeviceSynchronize();
  hipMemcpy(points, d_points, sizeof(double3) * out_width * out_height, hipMemcpyDeviceToHost);

  for(auto& object : obj_list) {
    /*
    int i_lb = constrain(object.x + object.w/2, 0, img_left.cols-1), 
    i_ub = i_lb + 1, 
    j_lb = constrain(object.y + object.h/2, 0, img_left.rows-1), 
    j_ub = j_lb + 1;
    */
    int i_lb = constrain(object.x, 0, img_left.cols-1), 
    i_ub = constrain(object.x + object.w, 0, img_left.cols-1), 
    j_lb = constrain(object.y, 0, img_left.rows-1), 
    j_ub = constrain(object.y + object.h, 0, img_left.rows-1);
    double X=0, Y=0, Z=0;

    #pragma omp parallel for //reduction(+:X,Y,Z)
    for(int i = i_lb; i < i_ub; i++) {
      for(int j = j_lb; j < j_ub; j++) {   
        X += points[j*out_width + i].x;  
        Y += points[j*out_width + i].y;     
        Z += points[j*out_width + i].z;  
      }
    } 
    //appendOBJECTS(X/((i_ub-i_lb)*(j_ub-j_lb)), Y/((i_ub-i_lb)*(j_ub-j_lb)), Z/((i_ub-i_lb)*(j_ub-j_lb)), object.r, object.g, object.b); 
    //if (draw_points == 1)
    appendOBJECTS(Y/((i_ub-i_lb)*(j_ub-j_lb)), -Z/((i_ub-i_lb)*(j_ub-j_lb)), X/((i_ub-i_lb)*(j_ub-j_lb)), object.r, object.g, object.b); 
  }
   
  if (!dmap.empty()) {
    // TODO : Do something
  }
  //updateGraph();
  end_timer(pc_t);
}

/*
 * Function:  generateDisparityMap 
 * --------------------
 * This function computes the dense disparity map using LIBELAS, and returns an 8-bit grayscale image Mat.
 * The disparity map is constructed with the left image as reference. The parameters for LIBELAS can be changed in the file src/elas/elas.h.
 * Any method other than LIBELAS can be implemented inside the generateDisparityMap function to generate disparity maps. One can use OpenCV’s StereoBM class as well. The output should be a 8-bit grayscale image.
 *
 *  Mat& left: The input left image
 *  Mat& right: The input right image
 *  returns: Mat output 8-bit grayscale image
 *
 */
Mat generateDisparityMap(Mat& left, Mat& right) {
  resetOBJECTS();
  if (left.empty() || right.empty()) 
    return left;
  const Size imsize = left.size();
  const int32_t dims[3] = {imsize.width, imsize.height, imsize.width};
  Mat leftdpf = Mat::zeros(imsize, CV_32F);
  Mat rightdpf = Mat::zeros(imsize, CV_32F);

  
  static Elas::parameters param(Elas::MIDDLEBURY);
  //Elas::parameters param(Elas::ROBOTICS);
  //Elas::parameters param;
  
  param.postprocess_only_left = true;
  //param.postprocess_only_left = false;
  
  //Elas elas(param);
  static ElasGPU elas(param);
  
  elas.process(left.data, right.data, leftdpf.ptr<float>(0), rightdpf.ptr<float>(0), dims);
  Mat dmap = Mat(out_img_size, CV_8UC1, Scalar(0));
  
  leftdpf.convertTo(dmap, CV_8UC1, 4.0);
  
  return dmap;
}

/*
 * Function:  imgCallback 
 * --------------------
 * Loads the input images into Mats
 * Undistorts and Rectifies the images with remap()
 * Generates disparity map with generateDisparityMap(img_left, img_right)
 * Displays output with imshow() and publishPointCloud()
 *
 *  const char* left_img_topic: path to left image
 *  const char* right_img_topic: path to right image
 *  returns: void
 *
 */

void imgCallback_video() {
  Mat left_img = left_img_OLD; Mat right_img = right_img_OLD;
  if (left_img.empty() || right_img.empty()){
    //printf("%s\n",left_img_topic);
    return;
  }

  Mat img_left, img_right, img_left_color_flip;

  cvtColor(left_img, img_left, COLOR_BGRA2GRAY);
  cvtColor(right_img, img_right, COLOR_BGRA2GRAY);

  //remap(tmpL, img_left, lmapx, lmapy, cv::INTER_LINEAR); remap(tmpR, img_right, rmapx, rmapy, cv::INTER_LINEAR);
  
  start_timer;   
  dmapOLD = generateDisparityMap(img_left, img_right);  
  end_timer(yd_t);
}

void imgCallback(const char* left_img_topic, const char* right_img_topic, int wait=0) {
printf("imgCalback called\n");
  Mat tmpL_Color = imread(left_img_topic, IMREAD_UNCHANGED);
  Mat tmpL = imread(left_img_topic, IMREAD_GRAYSCALE);
  Mat tmpR = imread(right_img_topic, IMREAD_GRAYSCALE);
  
  if (tmpL.empty() || tmpR.empty()) return;

  resize(tmpL_Color, tmpL_Color, out_img_size);
  resize(tmpL, tmpL, out_img_size);
  resize(tmpR, tmpR, out_img_size);

  Mat frame = tmpL_Color.clone();
  Mat img_left, img_right, img_left_color, img_left_color_flip;  
  
  img_left = tmpL; img_right = tmpR;

  //remap(tmpL, img_left, lmapx, lmapy, cv::INTER_LINEAR); remap(tmpR, img_right, rmapx, rmapy, cv::INTER_LINEAR);
  
  start_timer;
  auto f = std::async(std::launch::async, processYOLO, tmpL_Color); // Asynchronous call to YOLO
  Mat dmap = generateDisparityMap(img_left, img_right);  
  obj_list = f.get(); // Getting obj_list from the future object which the async call returns to f  
  end_timer(yd_t);

  publishPointCloud(frame, dmap);
  
  flip(tmpL_Color, img_left_color_flip,1);
  
  imshow("LEFT_C", img_left_color_flip);
}

/*
 * Function:  findRectificationMap 
 * --------------------
 * This function computes all the projection matrices and 
 * the rectification transformations using the stereoRectify 
 * and initUndistortRectifyMap functions respectively.
 *
 *  FileStorage& calib_file: The List in question
 *  Size finalSize: The data to tbe inserted
 *  returns: void
 *
 */
void findRectificationMap(FileStorage& calib_file, Size finalSize) {
  Rect validRoi[2];
  cout << "Starting rectification" << endl;

  /*
  void cv::stereoRectify  ( 
    InputArray  cameraMatrix1,
    InputArray  distCoeffs1,
    InputArray  cameraMatrix2,
    InputArray  distCoeffs2,
    Size        imageSize,
    InputArray  R,
    InputArray  T,
    OutputArray R1,
    OutputArray R2,
    OutputArray P1,
    OutputArray P2,
    OutputArray Q,
    int         flags = CALIB_ZERO_DISPARITY,
    double      alpha = -1,
    Size        newImageSize = Size(),
    Rect *      validPixROI1 = 0,
    Rect *      validPixROI2 = 0 
  )

  stereoRectify
  Computes rectification transforms for each head of a calibrated stereo camera.

  Paramers 
    cameraMatrix1   First camera intrinsic matrix.
    distCoeffs1     First camera distortion parameters.
    cameraMatrix2   Second camera intrinsic matrix.
    distCoeffs2     Second camera distortion parameters.
    imageSize       Size of the image used for stereo calibration.
    R               Rotation matrix from the coordinate system of the first camera to the second camera, see stereoCalibrate.
    T               Translation vector from the coordinate system of the first camera to the second camera, see stereoCalibrate.
    R1  Output 3x3  rectification transform (rotation matrix) for the first camera. This matrix brings points given in the unrectified first camera's coordinate system to points in the rectified first camera's coordinate system. In more technical terms, it performs a change of basis from the unrectified first camera's coordinate system to the rectified first camera's coordinate system.
    R2  Output 3x3  rectification transform (rotation matrix) for the second camera. This matrix brings points given in the unrectified second camera's coordinate system to points in the rectified second camera's coordinate system. In more technical terms, it performs a change of basis from the unrectified second camera's coordinate system to the rectified second camera's coordinate system.
    P1  Output 3x4  projection matrix in the new (rectified) coordinate systems for the first camera, i.e. it projects points given in the rectified first camera coordinate system into the rectified first camera's image.
    P2  Output 3x4  projection matrix in the new (rectified) coordinate systems for the second camera, i.e. it projects points given in the rectified first camera coordinate system into the rectified second camera's image.
    Q   Output 4×4  disparity-to-depth mapping matrix (see reprojectImageTo3D).
    flags           Operation flags that may be zero or CALIB_ZERO_DISPARITY . If the flag is set, the function makes the principal points of each camera have the same pixel coordinates in the rectified views. And if the flag is not set, the function may still shift the images in the horizontal or vertical direction (depending on the orientation of epipolar lines) to maximize the useful image area.
    alpha           Free scaling parameter. If it is -1 or absent, the function performs the default scaling. Otherwise, the parameter should be between 0 and 1. alpha=0 means that the rectified images are zoomed and shifted so that only valid pixels are visible (no black areas after rectification). alpha=1 means that the rectified image is decimated and shifted so that all the pixels from the original images from the cameras are retained in the rectified images (no source image pixels are lost). Any intermediate value yields an intermediate result between those two extreme cases.
    newImageSize    New image resolution after rectification. The same size should be passed to initUndistortRectifyMap (see the stereo_calib.cpp sample in OpenCV samples directory). When (0,0) is passed (default), it is set to the original imageSize . Setting it to a larger value can help you preserve details in the original image, especially when there is a big radial distortion.
    validPixROI1    Optional output rectangles inside the rectified images where all the pixels are valid. If alpha=0 , the ROIs cover the whole images. Otherwise, they are likely to be smaller (see the picture below).
    validPixROI2    Optional output rectangles inside the rectified images where all the pixels are valid. If alpha=0 , the ROIs cover the whole images. Otherwise, they are likely to be smaller (see the picture below).
  */
  //stereoRectify(K1, D1, K2, D2, calib_img_size, R, Mat(T), R1, R2, P1, P2, Q, CV_CALIB_ZERO_DISPARITY, 0, finalSize, &validRoi[0], &validRoi[1]);
  stereoRectify(K1, D1, K2, D2, calib_img_size, R, Mat(T), R1, R2, P1, P2, Q, 
                CALIB_ZERO_DISPARITY, 0, finalSize, &validRoi[0], &validRoi[1]);
  
  //P1 = (Mat_<double>(3,4) << 7.215377000000e+02, 0.000000000000e+00, 6.095593000000e+02, 4.485728000000e+01, 0.000000000000e+00, 7.215377000000e+02, 1.728540000000e+02, 2.163791000000e-01, 0.000000000000e+00, 0.000000000000e+00, 1.000000000000e+00, 2.745884000000e-03);
  //P2 = (Mat_<double>(3,4) << 7.215377000000e+02, 0.000000000000e+00, 6.095593000000e+02, -3.395242000000e+02, 0.000000000000e+00, 7.215377000000e+02, 1.728540000000e+02, 2.199936000000e+00, 0.000000000000e+00, 0.000000000000e+00, 1.000000000000e+00, 2.729905000000e-03);

  /*
  void cv::initUndistortRectifyMap  ( 
    InputArray  cameraMatrix,
    InputArray  distCoeffs,
    InputArray  R,
    InputArray  newCameraMatrix,
    Size  size,
    int   m1type,
    OutputArray   map1,
    OutputArray   map2 
  )

  initUndistortRectifyMap
    Computes the undistortion and rectification transformation map.
    The function computes the joint undistortion and rectification transformation and represents the result in the form of maps for remap. 
    The undistorted image looks like original, as if it is captured with a camera using the camera matrix =newCameraMatrix and zero distortion. 
    In case of a monocular camera, newCameraMatrix is usually equal to cameraMatrix, or it can be computed by getOptimalNewCameraMatrix for a better control over scaling. 
    In case of a stereo camera, newCameraMatrix is normally set to P1 or P2 computed by stereoRectify .
    Also, this new camera is oriented differently in the coordinate space, according to R. 
    That, for example, helps to align two heads of a stereo camera so that the epipolar lines on both images become horizontal and have the same y- coordinate (in case of a horizontally aligned stereo camera).

  Paramers 
    cameraMatrix    Input camera matrix A=[fx 0 cx; 0 fy cy; 0 0 1].
    distCoeffs      Input vector of distortion coefficients (k1,k2,p1,p2[,k3[,k4,k5,k6[,s1,s2,s3,s4[,τx,τy]]]]) of 4, 5, 8, 12 or 14 elements. If the vector is NULL/empty, the zero distortion coefficients are assumed.
    R Optional      rectification transformation in the object space (3x3 matrix). R1 or R2 , computed by stereoRectify can be passed here. If the matrix is empty, the identity transformation is assumed. In cvInitUndistortMap R assumed to be an identity matrix.
    newCameraMatrix New camera matrix A′=[f'x 0 c'x; 0 f'y c'y; 0 0 1].
    size            Undistorted image size.
    m1type          Type of the first output map that can be CV_32FC1, CV_32FC2 or CV_16SC2, see convertMaps
    map1            The first output map.
    map2            The second output map.
  */
  cv::initUndistortRectifyMap(K1, D1, R1, P1, finalSize, CV_32F, lmapx, lmapy);
  cv::initUndistortRectifyMap(K2, D2, R2, P2, finalSize, CV_32F, rmapx, rmapy);
  
  cout << "------------------" << endl;
  cout << "Done rectification" << endl;
  
}

void next(){
  static int iImage=0;
  if (video_mode){
    char left_img_topic[128], right_img_topic[128];
    size_t max_files = 465; // Just hardcoded the value for now

    Mat left_img, right_img, dmap, YOLOL_Color, img_left_color_flip;
    //thread th1(imgCallback_video);
    thread th1;

    play_video = 1;
    while (play_video){
      for(int iFrame = 0; iFrame < max_files; iFrame++){
        if (t_t!=0) printf("(FPS=%f) ", 1/t_t);
        
        start_timer;        
        strcpy(left_img_topic , format("%s/video/testing/image_02/%04d/%06d.png", kitti_path, iImage, iFrame).c_str());    
        strcpy(right_img_topic, format("%s/video/testing/image_03/%04d/%06d.png", kitti_path, iImage, iFrame).c_str());    

        left_img = imread(left_img_topic, IMREAD_UNCHANGED);
        right_img = imread(right_img_topic, IMREAD_UNCHANGED);
        resize(left_img, left_img, out_img_size);
        resize(right_img, right_img, out_img_size);        

        YOLOL_Color = left_img.clone();
        obj_list = processYOLO(YOLOL_Color);
        pred_list = get_predicted_boxes();
        append_old_objs(obj_list);
        obj_list.insert( obj_list.end(), pred_list.begin(), pred_list.end() );
        
        //auto f = std::async(std::launch::async, processYOLO, YOLOL_Color); // Asynchronous call to YOLO 

        if (iFrame%frame_skip == 0) {
          //printf("(DISP) \t ");
          //imgCallback_video(left_img, right_img, dmap);
          left_img_OLD = left_img.clone();
          right_img_OLD = right_img.clone();

          //disp_parallel = std::async(imgCallback_video);
          th1 = thread(imgCallback_video);
        }
          
        if (iFrame%frame_skip == frame_skip-1) {
          //printf("(JOIN) \t");
          th1.join();
          dmap = dmapOLD.clone();
        }

        printf("(%d, %d) ", dmap.rows, dmap.cols);
        
        //if (iFrame%frame_skip == frame_skip-1) {
        // th1.join();
        //  dmap = dmapOLD.clone();
        //}
    
        Mat rgba;
        cvtColor(left_img, rgba, cv::COLOR_BGR2BGRA);
        color = (uchar4*)rgba.ptr<unsigned char>(0);
        //obj_list = f.get(); // Getting obj_list from the future object which the async call return to f
        publishPointCloud(left_img, dmap);
        //printf("(PC Done) ");
        updateGraph();

        if (0){
          flip(left_img, img_left_color_flip,1);
          namedWindow("Detections", cv::WINDOW_NORMAL); // Needed to allow resizing of the image shown
          namedWindow("Disparity", cv::WINDOW_NORMAL); // Needed to allow resizing of the image shown
          imshow("Detections", YOLOL_Color);
          imshow("Disparity", dmap);
          waitKey(1);
        }
        end_timer(t_t);        
        printf("(t_t=%f, \t yd_t=%f, \t pc_t=%f)\n",t_t, yd_t, pc_t);
      }
    }
  } 
  else {
    printf("Next image\n");
    char left_img_topic[128], right_img_topic[128];
    strcpy(left_img_topic , format("%s/video/testing/image_2/0000/%06d.png", kitti_path, iImage).c_str());    
    strcpy(right_img_topic, format("%s/video/testing/image_3/0000/%06d.png", kitti_path, iImage).c_str());       
    imgCallback(left_img_topic, right_img_topic);
    iImage++;
  }
}

void next_video() {
  play_video = 0;
}

void imageLoop() {
  while (1) next();
}

int main(int argc, const char** argv){  
  initYOLO();

  static struct poptOption options[] = { 
    { "kitti_path",'k',POPT_ARG_STRING,&kitti_path,0,"Path to KITTI Dataset","STR" },
    { "video_mode",'v',POPT_ARG_INT,&video_mode,0,"Set v=1 Kitti video mode","NUM" },
    { "draw_points",'p',POPT_ARG_INT,&draw_points,0,"Set p=1 to plot out points","NUM" },
    { "frame_skip",'f',POPT_ARG_INT,&frame_skip,0,"Set frame_skip to skip disparity generation for f frames","NUM" },
    { "debug",'d',POPT_ARG_INT,&debug,0,"Set d=1 for cam to robot frame calibration","NUM" },
    POPT_AUTOHELP
    { NULL, 0, 0, NULL, 0, NULL, NULL }
  };

  poptContext poptCONT = poptGetContext("main", argc, argv, options, POPT_CONTEXT_KEEP_FIRST);
  int c = 0; while(c >= 0) c = poptGetNextOpt(poptCONT);

  printf("KITTI Path: %s \n", kitti_path);
  
  calib_img_size = Size(calib_width, calib_height);
  out_img_size = Size(out_width, out_height);
  
  calib_file = FileStorage(calib_file_name, FileStorage::READ);
  calib_file["K1"] >> K1;
  calib_file["K2"] >> K2;
  calib_file["D1"] >> D1;
  calib_file["D2"] >> D2;
  calib_file["R"]  >> R;
  calib_file["T"]  >> T;
  calib_file["XR"] >> XR;
  calib_file["XT"] >> XT;

 
  cout << " K1 : " << K1 << "\n D1 : " << D1 << "\n R1 : " << R1 << "\n P1 : " << P1  
       << "\n K2 : " << K2 << "\n D2 : " << D2 << "\n R2 : " << R2 << "\n P2 : " << P2 << '\n';
  
  findRectificationMap(calib_file, out_img_size);
  
  cudaInit();

  setCallback(next_video);
  thread th1(imageLoop);
  startGraphics(out_width, out_height);
  th1.join();
  clean();
  return 0;
}
